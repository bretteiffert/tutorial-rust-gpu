#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>


#include <chrono>

#define MAX_ERR 1e-6

// CUDA kernel
__global__ void vector_add(float *out, float *a, float *b, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Handling arbitrary vector size
    if (tid < n){
        out[tid] = a[tid] + b[tid];
    }
}

// helper C function called via Rust FFI to faciliate memory transfers and kernel launches
extern "C" void run_vector_add(float * a, float * b, float * out,  int N) { 

    auto start = std::chrono::steady_clock::now();
     	
    float *d_a, *d_b, *d_out;    

    // Allocate device memory
    hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);
    
    // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);
    
    // Executing kernel 
    int block_size = 256;
    int grid_size = ((N + block_size) / block_size);
    vector_add<<<grid_size,block_size>>>(d_out, d_a, d_b, N);
    
    auto end = std::chrono::steady_clock::now();

    printf("%d ms\n", std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count());

    // Transfer data back to host memory
    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

    // Deallocate device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    // Host memory freeing handled by Rust
}
